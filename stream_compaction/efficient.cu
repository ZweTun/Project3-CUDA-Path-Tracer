#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"
#include <>

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }


        #define blockSize 128
        int* obuffer;
        int* ibuffer;


        __global__ void upSweep(int n, int* idata, int layer) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);

			int skip = 1 << (layer + 1); // powf(2, layer + 1);

            int i = (index) * skip;
            if (i + skip - 1 >= n) {
                return;
            }
          
           
            idata[int(i + skip - 1)] += idata[int(i + (skip >> 1) - 1)];

        
        }


        __global__ void downSweep(int n, int* idata, int layer) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
          
   
			int skip = 1 << (layer + 1); // powf(2, layer + 1);
            int i = index * skip;
            if (i + skip - 1 >= n) {
                return;
			}   
          


            int t = idata[int(i + (skip >> 1) - 1)];
            
        
            idata[int(i + (skip >> 1) - 1)] = idata[int(i + skip - 1)];
            idata[int(i + skip - 1)] += t;
		}

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
           

            int size = 1 << ilog2ceil(n); 
       
            //Init Buffers 
            int* obuffer;
            int* ibuffer;
            hipMalloc((void**)&obuffer, size * sizeof(int));
            hipMalloc((void**)&ibuffer, size * sizeof(int));
            hipMemset(ibuffer, 0, (size) * sizeof(int));

            hipMemcpy(obuffer, odata, size * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(ibuffer, idata, size * sizeof(int), hipMemcpyHostToDevice);
          
            hipDeviceSynchronize();
          
			// up sweep
            int numBlocks;


            for (int layer = 0; layer <= ilog2ceil(size) - 1; layer++) {
                int numThreads = size / int(powf(2, layer + 1));
				numBlocks = (numThreads + blockSize - 1) / blockSize;
                upSweep<<<numBlocks, blockSize>>>(size, ibuffer, layer);
                hipDeviceSynchronize();
               
             
            }

	        //Set ibuffer[n - 1] = 0 
			hipMemset(ibuffer + size - 1, 0, sizeof(int));
         
         

           // down sweep 
            for (int layer = ilog2ceil(size) - 1; layer >= 0; layer--) {
                int numThreads = size / int(powf(2, layer + 1));
      

                numBlocks = (numThreads + blockSize - 1) / blockSize;
                downSweep<<<numBlocks, blockSize>>>(size, ibuffer, layer);
                hipDeviceSynchronize();
               

			}
            timer().endGpuTimer();



   
            hipMemcpy(odata, ibuffer, n * sizeof(int), hipMemcpyDeviceToHost);
         
         
		
			hipFree(ibuffer);
			hipFree(obuffer);
            
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {

          //  timer().startGpuTimer();
			//Init Buffers
            int* boolBufferDevice; 
			int* obuffer;
            int* ibuffer;
			int* scanBuffer = new int[n];
			int* scanBufferDevice;

			hipMalloc((void**)&scanBufferDevice, n * sizeof(int));
            hipMalloc((void**)&boolBufferDevice, n * sizeof(int));
            hipMalloc((void**)&ibuffer, n * sizeof(int));
            hipMalloc((void**)&obuffer, n * sizeof(int));

            hipMemcpy(ibuffer, idata, n * sizeof(int), hipMemcpyHostToDevice);
           
            int numBlocks = (n + blockSize - 1) / blockSize;
            dim3 fullBlocksPerGrid(numBlocks);
            //Map to boolean 
            Common::kernMapToBoolean<<<fullBlocksPerGrid, blockSize>>>(n, boolBufferDevice, ibuffer);
           
			int* boolBuffer = new int[n];
            hipMemcpy(boolBuffer, boolBufferDevice, n * sizeof(int), hipMemcpyDeviceToHost);

          
            //Scan boolean array 
            scan(n, scanBuffer, boolBuffer);
   

            hipMemcpy(scanBufferDevice, scanBuffer, n * sizeof(int), hipMemcpyHostToDevice);
			
            Common::kernScatter <<<fullBlocksPerGrid, blockSize >> > (n, obuffer, ibuffer, boolBufferDevice, scanBufferDevice);
            ////Scatter results 
       
         
            //  timer().endGpuTimer();

			//Get count by looking at last element of scan + last element of bool
			//Last element of scan holds number of elements up to n-1, so 
            //add bool[n-1] to get the full count
            int count;
            hipMemcpy(&count, scanBufferDevice + n - 1, sizeof(int), hipMemcpyDeviceToHost);
            int lastBool;
            hipMemcpy(&lastBool, boolBufferDevice + n - 1, sizeof(int), hipMemcpyDeviceToHost);
            count += lastBool;
            hipFree(boolBuffer);
            hipFree(scanBuffer);
            hipMemcpy(odata, obuffer, n * sizeof(int), hipMemcpyDeviceToHost);

		
         
			return count;

       
           
        }
    }
}
